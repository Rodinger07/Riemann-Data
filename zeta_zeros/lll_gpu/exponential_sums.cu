#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

__global__ void compute_exponential_sums(hipfftDoubleComplex* result, const double* t_values, int n, double* reduced_basis, int basis_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        double t = t_values[idx];
        double real_sum = 0.0;
        double imag_sum = 0.0;
        
        for (int k = 0; k < basis_size; k++) {
            double phase = t * reduced_basis[k];
            real_sum += cos(phase);
            imag_sum += sin(phase);
        }
        
        result[idx].x = real_sum;
        result[idx].y = imag_sum;
    }
}

void gpu_exponential_sums(const double* t_values, int n, double* reduced_basis, int basis_size, hipfftDoubleComplex* result) {
    double* d_t;
    double* d_basis;
    hipfftDoubleComplex* d_result;
    
    hipMalloc(&d_t, n * sizeof(double));
    hipMalloc(&d_basis, basis_size * sizeof(double));
    hipMalloc(&d_result, n * sizeof(hipfftDoubleComplex));
    
    hipMemcpy(d_t, t_values, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_basis, reduced_basis, basis_size * sizeof(double), hipMemcpyHostToDevice);
    
    dim3 blockDim(256);
    dim3 gridDim((n + blockDim.x - 1) / blockDim.x);
    
    compute_exponential_sums<<<gridDim, blockDim>>>(d_result, d_t, n, d_basis, basis_size);
    
    hipMemcpy(result, d_result, n * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);
    
    hipFree(d_t);
    hipFree(d_basis);
    hipFree(d_result);
}
