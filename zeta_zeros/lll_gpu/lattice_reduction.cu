#include <hip/hip_runtime.h>
#include <fplll.h>

extern "C" void lll_reduction(double** basis, int n, int m, double** reduced_basis) {
    ZZ_mat<mpz_t> matrix(n, m);
    
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            matrix[i][j] = basis[i][j] * 1e15;
        }
    }
    
    lll_reduction(matrix, LLL_DEF_DELTA, LLL_DEF_ETA, LM_WRAPPER, FT_DEFAULT, 0, LLL_DEFAULT);
    
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            reduced_basis[i][j] = to_double(matrix[i][j]) / 1e15;
        }
    }
}
